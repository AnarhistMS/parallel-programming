
#include <hip/hip_runtime.h>
__device__
unsigned char GetPixel(unsigned char * src, unsigned int w, unsigned int h, int x, int y, int c)
{
	x = x < 0 ? 0 : x;
	x = x < w ? x : w - 1;
	y = y < 0 ? 0 : y;
	y = y < h ? y : h - 1;
	return src[3 * (w * y + x) + c];
}

__global__
void grayscale_kernel(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < h && x < w)
	{
		float luma = 0;
		luma += 0.2126f * src[3 * (w * y + x) + 0];
		luma += 0.7152f * src[3 * (w * y + x) + 1];
		luma += 0.0722f * src[3 * (w * y + x) + 2];
		
		unsigned char l = (unsigned char)luma;
		
		dest[3 * (w * y + x) + 0] = l;
		dest[3 * (w * y + x) + 1] = l;
		dest[3 * (w * y + x) + 2] = l;
	}
}

__global__
void sobel_kernel(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.z;

	if (x < w && y < h)
	{
		/*
			Sobel X-filter:             Sobel Y-filter:
			-1   0  +1                  -1  -2  -1
			-2   0  +2                   0   0   0
			-1   0  +1                  +1  +2  +1
		*/
		float Gx = 0;
		float Gy = 0;

		if (x > 1)
		{
			if (y > 1)
				Gx -= src[3 * (w * (y - 1) + (x - 1)) + c];
			Gx -= 2.0f * src[3 * (w * y + (x - 1)) + c];
			if (y < h - 1)
				Gx -= src[3 * (w * (y + 1) + (x - 1)) + c];
		}

		if (x < w - 1)
		{
			if (y > 1)
				Gx += src[3 * (w * (y - 1) + (x + 1)) + c];
			Gx += 2.0f * src[3 * (w * y + (x + 1)) + c];
			if (y < h - 1)
				Gx += src[3 * (w * (y + 1) + (x + 1)) + c];
		}

		if (y > 1)
		{
			if (x > 1)
				Gy -= src[3 * (w * (y - 1) + (x - 1)) + c];
			Gx -= 2.0f * src[3 * (w * (y - 1) + x) + c];
			if (x < w - 1)
				Gx -= src[3 * (w * (y - 1) + (x + 1)) + c];

		}

		if (y < h - 1)
		{
			if (x > 1)
				Gy += src[3 * (w * (y + 1) + (x - 1)) + c];
			Gx += 2.0f * src[3 * (w * (y + 1) + x) + c];
			if (x < w - 1)
				Gx += src[3 * (w * (y + 1) + (x + 1)) + c];
		}

		float G = sqrt(Gx*Gx + Gy*Gy);
		dest[3 * (w * y + x) + c] = G > 32 ? 255 : 0;
	}
}

__global__
void sobel_kernel_shared(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	int x_block = blockIdx.x * blockDim.x;
	int y_block = blockIdx.y * blockDim.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int c = blockIdx.z;

	__shared__ unsigned char pixels[16 + 2][16 + 2];
	for (unsigned int i = blockDim.x * ty + tx;
		i < (blockDim.x + 2) * (blockDim.y + 2);
		i += blockDim.x * blockDim.y)
	{
		char y_off = i / (blockDim.x + 2);
		char x_off = i % (blockDim.x + 2);
		pixels[y_off][x_off] = GetPixel(src, w, h, x_block + x_off - 1, y_block + y_off - 1, c);
	}

	__syncthreads();

	if (x_block + tx < w && y_block + ty < h)
	{
		/*
		Sobel X-filter:             Sobel Y-filter:
		-1   0  +1                  -1  -2  -1
		-2   0  +2                   0   0   0
		-1   0  +1                  +1  +2  +1
		*/

		float Gx = 0
			- 1.0 * pixels[ty + 0][tx + 0]
			- 2.0 * pixels[ty + 1][tx + 0]
			- 1.0 * pixels[ty + 2][tx + 0]
			+ 1.0 * pixels[ty + 0][tx + 2]
			+ 2.0 * pixels[ty + 1][tx + 2]
			+ 1.0 * pixels[ty + 2][tx + 2];

		float Gy = 0
			- 1.0 * pixels[ty + 0][tx + 0]
			- 2.0 * pixels[ty + 0][tx + 1]
			- 1.0 * pixels[ty + 0][tx + 2]
			+ 1.0 * pixels[ty + 2][tx + 0]
			+ 2.0 * pixels[ty + 2][tx + 1]
			+ 1.0 * pixels[ty + 2][tx + 2];

		float G = sqrt(Gx*Gx + Gy*Gy);
		dest[3 * (w * (y_block + ty) + x_block + tx) + c] = G > 32 ? 255 : 0;
	}
}

extern "C"
void GPUFiltering(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	// �������������� � ������� ������
	dim3 BlockDim1(16, 16, 1);
	dim3 GridDim1((w - 1)/BlockDim1.x + 1, (h - 1)/BlockDim1.y + 1, 1);
	grayscale_kernel<<<GridDim1, BlockDim1>>>(src, dest, w, h);

	// ������ ������
	dim3 BlockDim2(16, 16, 1);
	dim3 GridDim2((w - 1) / BlockDim2.x + 1, (h - 1) / BlockDim2.y + 1, 3);
	sobel_kernel_shared<<<GridDim2, BlockDim2>>>(dest, src, w, h);
}
