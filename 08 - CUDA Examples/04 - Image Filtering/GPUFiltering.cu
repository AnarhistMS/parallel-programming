
#include <hip/hip_runtime.h>
﻿const unsigned char BLOCK_DIM_X = 16;
const unsigned char BLOCK_DIM_Y = 16;

__device__
unsigned char GetPixel(unsigned char * src, unsigned int w, unsigned int h, int x, int y, int c)
{
	x = x < 0 ? 0 : x;
	x = x < w ? x : w - 1;
	y = y < 0 ? 0 : y;
	y = y < h ? y : h - 1;
	return src[3 * (w * y + x) + c];
}

__global__
void grayscale_kernel(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < h && x < w)
	{
		float luma = 0
			+ 0.2126f * GetPixel(src, w, h, x, y, 0)
			+ 0.7152f * GetPixel(src, w, h, x, y, 1)
			+ 0.0722f * GetPixel(src, w, h, x, y, 2);
		
		unsigned char l = (unsigned char)luma;
		dest[3 * (w * y + x) + 0] = l;
		dest[3 * (w * y + x) + 1] = l;
		dest[3 * (w * y + x) + 2] = l;
	}
}

__global__
void sobel_kernel(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.z;

	if (x < w && y < h)
	{
		/*
			Sobel X-filter:             Sobel Y-filter:
			-1   0  +1                  -1  -2  -1
			-2   0  +2                   0   0   0
			-1   0  +1                  +1  +2  +1
		*/
		float Gx = 0
			- 1.0 * GetPixel(src, w, h, x - 1, y - 1, c)
			- 2.0 * GetPixel(src, w, h, x - 1, y + 0, c)
			- 1.0 * GetPixel(src, w, h, x - 1, y + 1, c)
			+ 1.0 * GetPixel(src, w, h, x + 1, y - 1, c)
			+ 2.0 * GetPixel(src, w, h, x + 1, y + 0, c)
			+ 1.0 * GetPixel(src, w, h, x + 1, y + 1, c);

		float Gy = 0
			- 1.0 * GetPixel(src, w, h, x - 1, y - 1, c)
			- 2.0 * GetPixel(src, w, h, x + 0, y - 1, c)
			- 1.0 * GetPixel(src, w, h, x + 1, y - 1, c)
			+ 1.0 * GetPixel(src, w, h, x - 1, y + 1, c)
			+ 2.0 * GetPixel(src, w, h, x + 0, y + 1, c)
			+ 1.0 * GetPixel(src, w, h, x + 1, y + 1, c);

		float G = sqrt(Gx*Gx + Gy*Gy);
		dest[3 * (w * y + x) + c] = G > 32 ? 255 : 0;
	}
}

__global__
void sobel_kernel_shared(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	int x_block = blockIdx.x * blockDim.x;
	int y_block = blockIdx.y * blockDim.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int c = blockIdx.z;

	__shared__ unsigned char pixels[BLOCK_DIM_Y + 2][BLOCK_DIM_X + 2];
	for (unsigned int i = blockDim.x * ty + tx;
		i < (blockDim.x + 2) * (blockDim.y + 2);
		i += blockDim.x * blockDim.y)
	{
		char y_off = i / (blockDim.x + 2);
		char x_off = i % (blockDim.x + 2);
		pixels[y_off][x_off] = GetPixel(src, w, h, x_block + x_off - 1, y_block + y_off - 1, c);
	}

	__syncthreads();

	if (x_block + tx < w && y_block + ty < h)
	{
		/*
		Sobel X-filter:             Sobel Y-filter:
		-1   0  +1                  -1  -2  -1
		-2   0  +2                   0   0   0
		-1   0  +1                  +1  +2  +1
		*/

		float Gx = 0
			- 1.0 * pixels[ty + 0][tx + 0]
			- 2.0 * pixels[ty + 1][tx + 0]
			- 1.0 * pixels[ty + 2][tx + 0]
			+ 1.0 * pixels[ty + 0][tx + 2]
			+ 2.0 * pixels[ty + 1][tx + 2]
			+ 1.0 * pixels[ty + 2][tx + 2];

		float Gy = 0
			- 1.0 * pixels[ty + 0][tx + 0]
			- 2.0 * pixels[ty + 0][tx + 1]
			- 1.0 * pixels[ty + 0][tx + 2]
			+ 1.0 * pixels[ty + 2][tx + 0]
			+ 2.0 * pixels[ty + 2][tx + 1]
			+ 1.0 * pixels[ty + 2][tx + 2];

		float G = sqrt(Gx*Gx + Gy*Gy);
		dest[3 * (w * (y_block + ty) + x_block + tx) + c] = G > 32 ? 255 : 0;
	}
}

extern "C"
void GPUFiltering(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	// преобразование в оттенки серого
	dim3 BlockDim1(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
	dim3 GridDim1((w - 1)/BlockDim1.x + 1, (h - 1)/BlockDim1.y + 1, 1);
	grayscale_kernel<<<GridDim1, BlockDim1>>>(src, src, w, h);

	// фильтр Собеля
	dim3 BlockDim2(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
	dim3 GridDim2((w - 1) / BlockDim2.x + 1, (h - 1) / BlockDim2.y + 1, 3);
	sobel_kernel<<<GridDim2, BlockDim2>>>(src, dest, w, h);
}

extern "C"
void GPUFilteringShared(unsigned char * src, unsigned char * dest, unsigned int w, unsigned int h)
{
	// преобразование в оттенки серого
	dim3 BlockDim1(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
	dim3 GridDim1((w - 1) / BlockDim1.x + 1, (h - 1) / BlockDim1.y + 1, 1);
	grayscale_kernel <<<GridDim1, BlockDim1 >>>(src, src, w, h);

	// фильтр Собеля
	dim3 BlockDim2(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
	dim3 GridDim2((w - 1) / BlockDim2.x + 1, (h - 1) / BlockDim2.y + 1, 3);
	sobel_kernel_shared<<<GridDim2, BlockDim2 >>>(src, dest, w, h);
}
